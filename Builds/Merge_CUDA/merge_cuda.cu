#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <cmath>
#include <stdio.h>
#include <stdlib.h>

#include <caliper/cali.h>
#include <caliper/cali-manager.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <adiak.hpp>

using namespace std;

int THREADS;
int BLOCKS;
int NUM_VALS;

const char *data_init = "data_init";
const char *data_init_h2d = "data_init_h2d";
const char *data_init_d2h = "data_init_d2h";
const char *comp = "comp";
const char *comp_large = "comp_large";
const char *comm = "comm";
const char *comm_large = "comm_large";
const char *comp_h2d = "comp_h2d";
const char *comp_d2h = "comp_d2h";
const char *correctness_check = "correctness_check";
const char *correctness_h2d = "correctness_h2d";
const char *correctness_d2h = "correctness_d2h";

void printArray(float *arr, int size)
{
    for (int i = 0; i < size; i++)
    {
        printf("%f ", arr[i]);
    }
    printf("\n");
}

__global__ void random_fill(float *nums, int size, const char *input_type)
{
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    hiprandState state;
    hiprand_init(1, index, 0, &state);

    nums[index] = (float)hiprand_uniform(&state) * size;
}

__global__ void sorted_fill(float *nums, int size, const char *input_type)
{
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    nums[index] = (float)index;
}

__global__ void reverse_fill(float *nums, int size, const char *input_type)
{
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    nums[index] = (float)(size - index - 1);
}

__global__ void nearly_fill(float *nums, int size, const char *input_type)
{
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    hiprandState state;
    hiprand_init(1, index, 0, &state);

    nums[index] = (float)hiprand_uniform(&state) * blockIdx.x;
}

void fill_array(float *nums, const char *input_type)
{
    float *dev_nums;
    size_t size = NUM_VALS * sizeof(float);

    hipMalloc((void **)&dev_nums, size);

    // MEM COPY FROM HOST TO DEVICE
    CALI_MARK_BEGIN(comm);
    CALI_MARK_BEGIN(comm_large);
    CALI_MARK_BEGIN(data_init_h2d);
    hipMemcpy(dev_nums, nums, size, hipMemcpyHostToDevice);
    CALI_MARK_END(data_init_h2d);
    CALI_MARK_END(comm_large);
    CALI_MARK_END(comm);

    dim3 blocks(BLOCKS, 1);
    dim3 threads(THREADS, 1);

    // FILLING ARRAY
    CALI_MARK_BEGIN(data_init);
    if (strcmp(input_type, "random") == 0)
    {
        random_fill<<<blocks, threads>>>(dev_nums, NUM_VALS, input_type);
    }
    if (strcmp(input_type, "sorted") == 0)
    {
        sorted_fill<<<blocks, threads>>>(dev_nums, NUM_VALS, input_type);
    }
    if (strcmp(input_type, "reverse") == 0)
    {
        reverse_fill<<<blocks, threads>>>(dev_nums, NUM_VALS, input_type);
    }
    if (strcmp(input_type, "nearly") == 0)
    {
        nearly_fill<<<blocks, threads>>>(dev_nums, NUM_VALS, input_type);
    }
    CALI_MARK_END(data_init);

    // MEM COPY FROM DEVICE TO HOST
    CALI_MARK_BEGIN(comm);
    CALI_MARK_BEGIN(comm_large);
    CALI_MARK_BEGIN(data_init_d2h);
    hipMemcpy(nums, dev_nums, size, hipMemcpyDeviceToHost);
    CALI_MARK_END(data_init_d2h);
    CALI_MARK_END(comm_large);
    CALI_MARK_END(comm);

    hipFree(dev_nums);
}

__global__ void confirm_sorted_step(float *nums, int size, bool *sorted)
{
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    if (index < size - 1)
    {
        if (nums[index] > nums[index + 1])
        {
            *sorted = false;
        }
    }
}

bool confirm_sorted(float *nums)
{
    float *dev_nums;
    bool *dev_sorted;
    bool sorted = true;
    size_t size = NUM_VALS * sizeof(float);

    hipMalloc((void **)&dev_nums, size);
    hipMalloc((void **)&dev_sorted, sizeof(bool));

    // MEM COPY FROM HOST TO DEVICE
    CALI_MARK_BEGIN(comm);
    CALI_MARK_BEGIN(comm_large);
    CALI_MARK_BEGIN(correctness_h2d);
    hipMemcpy(dev_nums, nums, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_sorted, &sorted, sizeof(bool), hipMemcpyHostToDevice);
    CALI_MARK_END(correctness_h2d);
    CALI_MARK_END(comm_large);
    CALI_MARK_END(comm);

    dim3 blocks(BLOCKS, 1);
    dim3 threads(THREADS, 1);

    // CHECKING CORRECTNESS
    CALI_MARK_BEGIN(correctness_check);
    confirm_sorted_step<<<blocks, threads>>>(dev_nums, NUM_VALS, dev_sorted);
    hipDeviceSynchronize();
    CALI_MARK_END(correctness_check);

    // MEM COPY FROM DEVICE TO HOST
    CALI_MARK_BEGIN(comm);
    CALI_MARK_BEGIN(comm_large);
    CALI_MARK_BEGIN(correctness_d2h);
    hipMemcpy(&sorted, dev_sorted, sizeof(bool), hipMemcpyDeviceToHost);
    CALI_MARK_END(correctness_d2h);
    CALI_MARK_END(comm_large);
    CALI_MARK_END(comm);

    hipFree(dev_nums);
    hipFree(dev_sorted);
    return sorted;
}

void mergeHost(float *array, int const left, int const mid,
               int const right)
{
    int const subArrayOne = mid - left + 1;
    int const subArrayTwo = right - mid;

    // Create temp arrays
    auto *leftArray = new float[subArrayOne],
         *rightArray = new float[subArrayTwo];

    // Copy data to temp arrays leftArray[] and rightArray[]
    for (auto i = 0; i < subArrayOne; i++)
        leftArray[i] = array[left + i];
    for (auto j = 0; j < subArrayTwo; j++)
        rightArray[j] = array[mid + 1 + j];

    auto indexOfSubArrayOne = 0, indexOfSubArrayTwo = 0;
    int indexOfMergedArray = left;

    // Merge the temp arrays back into array[left..right]
    while (indexOfSubArrayOne < subArrayOne && indexOfSubArrayTwo < subArrayTwo)
    {
        if (leftArray[indexOfSubArrayOne] <= rightArray[indexOfSubArrayTwo])
        {
            array[indexOfMergedArray] = leftArray[indexOfSubArrayOne];
            indexOfSubArrayOne++;
        }
        else
        {
            array[indexOfMergedArray] = rightArray[indexOfSubArrayTwo];
            indexOfSubArrayTwo++;
        }
        indexOfMergedArray++;
    }

    // Copy the remaining elements of
    // left[], if there are any
    while (indexOfSubArrayOne < subArrayOne)
    {
        array[indexOfMergedArray] = leftArray[indexOfSubArrayOne];
        indexOfSubArrayOne++;
        indexOfMergedArray++;
    }

    // Copy the remaining elements of
    // right[], if there are any
    while (indexOfSubArrayTwo < subArrayTwo)
    {
        array[indexOfMergedArray] = rightArray[indexOfSubArrayTwo];
        indexOfSubArrayTwo++;
        indexOfMergedArray++;
    }
    delete[] leftArray;
    delete[] rightArray;
}

__device__ void merge(float *data, float *temp, int left, int mid, int right)
{
    int i = left;
    int j = mid + 1;
    int k = left;

    while (i <= mid && j <= right)
    {
        if (data[i] <= data[j])
        {
            temp[k] = data[i];
            i++;
        }
        else
        {
            temp[k] = data[j];
            j++;
        }
        k++;
    }

    while (i <= mid)
    {
        temp[k] = data[i];
        i++;
        k++;
    }

    while (j <= right)
    {
        temp[k] = data[j];
        j++;
        k++;
    }

    for (int x = left; x <= right; x++)
    {
        data[x] = temp[x];
    }
}

__global__ void mergeSort(float *data, float *temp, int n, int chunkSize)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    // Calculate the range of elements to be sorted in this launch
    int start = tid * chunkSize;
    int end = min(start + chunkSize - 1, n - 1);

    // Perform merge sort for the specified range
    for (int currSize = 1; currSize < n; currSize *= 2)
    {
        for (int left = start; left < end; left += 2 * currSize)
        {
            int mid = min(left + currSize - 1, end);
            int right = min(left + 2 * currSize - 1, end);
            merge(data, temp, left, mid, right);
        }
        __syncthreads(); // Synchronize threads before the next iteration
    }
}

int main(int argc, char *argv[])
{
    CALI_CXX_MARK_FUNCTION;
    cali::ConfigManager mgr;
    mgr.start();

    // retrieve user input
    const char *input_type = argv[1];
    THREADS = atoi(argv[2]);
    NUM_VALS = atoi(argv[3]);
    BLOCKS = NUM_VALS / THREADS;

    printf("Number of threads: %d\n", THREADS);
    printf("Number of values: %d\n", NUM_VALS);
    printf("Number of blocks: %d\n\n", BLOCKS);

    // initialize arrays
    float *h_data = (float *)malloc(sizeof(float) * NUM_VALS); // Host array
    float *d_data, *d_temp;                                    // Device arrays

    // fill array
    fill_array(h_data, input_type);
    cout << "Data Initialized" << endl;

    // Allocate memory on the device
    hipMalloc((void **)&d_data, sizeof(float) * NUM_VALS);
    hipMalloc((void **)&d_temp, sizeof(float) * NUM_VALS);

    // Copy data from the host to the device
    CALI_MARK_BEGIN(comm);
    CALI_MARK_BEGIN(comm_large);
    CALI_MARK_BEGIN(comp_h2d);
    hipMemcpy(d_data, h_data, sizeof(float) * NUM_VALS, hipMemcpyHostToDevice);
    CALI_MARK_END(comp_h2d);
    CALI_MARK_END(comm_large);
    CALI_MARK_END(comm);

    // Launch the CUDA kernel to perform merge sort
    CALI_MARK_BEGIN(comp);
    CALI_MARK_BEGIN(comp_large);
    int chunkSize = 256; // Adjust the chunk size as needed

    for (int i = 0; i < NUM_VALS / chunkSize; ++i)
    {
        mergeSort<<<BLOCKS, THREADS>>>(d_data, d_temp, NUM_VALS, chunkSize);
        hipDeviceSynchronize();
    }
    CALI_MARK_END(comp_large);
    CALI_MARK_END(comp);

    // Copy the sorted data back to the host
    CALI_MARK_BEGIN(comm);
    CALI_MARK_BEGIN(comm_large);
    CALI_MARK_BEGIN(comp_d2h);
    hipMemcpy(h_data, d_data, sizeof(float) * NUM_VALS, hipMemcpyDeviceToHost);
    CALI_MARK_END(comp_d2h);
    CALI_MARK_END(comm_large);
    CALI_MARK_END(comm);

    CALI_MARK_BEGIN(comp);
    CALI_MARK_BEGIN(comp_large);
    // Perform the final merge to combine all chunks
    for (int i = 1; i < NUM_VALS / chunkSize; ++i)
    {
        mergeHost(h_data, 0, i * chunkSize - 1, std::min(((i + 1) * chunkSize - 1), NUM_VALS - 1));
    }
    CALI_MARK_END(comp_large);
    CALI_MARK_END(comp);

    // Clean up and free memory
    hipFree(d_data);
    hipFree(d_temp);

    // check correctness
    if (confirm_sorted(h_data))
    {
        cout << "Correctness Check Passed!" << endl;
    }
    else
    {
        cout << "Correctness Check Failed..." << endl;
    }
    // printArray(h_data, NUM_VALS);
    free(h_data);
    return 0;
}
