#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <cmath>
#include <stdio.h>
#include <stdlib.h>

#include <caliper/cali.h>
#include <caliper/cali-manager.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <adiak.hpp>

using namespace std;

int THREADS;
int BLOCKS;
int NUM_VALS;

const char* data_init = "data_init";
const char* data_gen_h2d = "data_gen_h2d";
const char* data_gen_d2h = "data_gen_d2h";
const char* comp = "comp";
const char* comp_large = "comp_large";
const char* comm = "comm";
const char* comp_h2d = "comp_h2d";
const char* comp_d2h = "comp_d2h";
const char* correctness_check = "correctness_check";
const char* correctness_h2d = "correctness_h2d";
const char* correctness_d2h = "correctness_d2h";


__global__ void random_fill(float* nums, int size, const char* input_type) {
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    hiprandState state;
    hiprand_init(1, index, 0, &state);

    nums[index] = (float)hiprand_uniform(&state) * size;
}

__global__ void sorted_fill(float* nums, int size, const char* input_type) {
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    nums[index] = (float)index;
}

__global__ void reverse_fill(float* nums, int size, const char* input_type) {
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    nums[index] = (float)(size - index - 1);
}

__global__ void nearly_fill(float* nums, int size, const char* input_type) {
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    hiprandState state;
    hiprand_init(1, index, 0, &state);

    nums[index] = (float)hiprand_uniform(&state) * blockIdx.x;
}

__global__ void bubble_sort_step(float* nums, int size, int oddeven) {
    int index = threadIdx.x + blockDim.x * blockIdx.x;

    // even step
    if(oddeven == 0 && (index * 2 + 1) < size){
		if(nums[index * 2] > nums[index * 2 + 1]){
			float temp = nums[index * 2];
			nums[index * 2] = nums[index * 2 + 1];
			nums[index * 2 + 1] = temp;
		}
	}

    // odd step
	if(oddeven == 1 && (index * 2 + 2) < size){
		if(nums[index * 2 + 1] > nums[index * 2 + 2]){
			float temp = nums[index * 2 + 1];
			nums[index * 2 + 1] = nums[index * 2 + 2];
			nums[index * 2 + 2] = temp;
		}
	}
}

__global__ void confirm_sorted_step(float* nums, int size, bool* sorted) {
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    if(index < size - 1) {
        if(nums[index] > nums[index + 1]) {
            *sorted = false;
        }
    }
}

void fill_array(float* nums, const char* input_type) {
    float *dev_nums;
    size_t size = NUM_VALS * sizeof(float);
    
    hipMalloc((void**) &dev_nums, size);

    //MEM COPY FROM HOST TO DEVICE
    CALI_MARK_BEGIN(comm);
    CALI_MARK_BEGIN(data_gen_h2d);
    hipMemcpy(dev_nums, nums, size, hipMemcpyHostToDevice);
    CALI_MARK_END(data_gen_h2d);
    CALI_MARK_END(comm);

    dim3 blocks(BLOCKS, 1);
    dim3 threads(THREADS, 1);

    // FILLING ARRAY
    CALI_MARK_BEGIN(data_init);
    if(strcmp(input_type, "random") == 0) {
        random_fill<<<blocks, threads>>>(dev_nums, NUM_VALS, input_type);
    }
    if(strcmp(input_type, "sorted") == 0) {
        sorted_fill<<<blocks, threads>>>(dev_nums, NUM_VALS, input_type);
    }
    if(strcmp(input_type, "reverse") == 0) {
        reverse_fill<<<blocks, threads>>>(dev_nums, NUM_VALS, input_type);
    }
    if(strcmp(input_type, "nearly") == 0) {
        nearly_fill<<<blocks, threads>>>(dev_nums, NUM_VALS, input_type);
    }
    CALI_MARK_END(data_init);

    //MEM COPY FROM DEVICE TO HOST
    CALI_MARK_BEGIN(comm);
    CALI_MARK_BEGIN(data_gen_d2h);
    hipMemcpy(nums, dev_nums, size, hipMemcpyDeviceToHost);
    CALI_MARK_END(data_gen_d2h);
    CALI_MARK_END(comm);

    hipFree(dev_nums);
}

void bubble_sort(float* nums) {
    float *dev_nums;
    size_t size = NUM_VALS * sizeof(float);
    
    hipMalloc((void**) &dev_nums, size);

    //MEM COPY FROM HOST TO DEVICE
    CALI_MARK_BEGIN(comm);
    CALI_MARK_BEGIN(comp_h2d);
    hipMemcpy(dev_nums, nums, size, hipMemcpyHostToDevice);
    CALI_MARK_END(comp_h2d);
    CALI_MARK_END(comm);

    dim3 blocks(BLOCKS, 1);
    dim3 threads(THREADS, 1);

    // ODD-EVEN BUBBLE SORT
    CALI_MARK_BEGIN(comp);
    CALI_MARK_BEGIN(comp_large);
    for(int i = 0; i < NUM_VALS; i++) {
        bubble_sort_step<<<blocks, threads>>>(dev_nums, NUM_VALS, i % 2);
    }
    CALI_MARK_END(comp_large);
    CALI_MARK_END(comp);

    //MEM COPY FROM DEVICE TO HOST
    CALI_MARK_BEGIN(comm);
    CALI_MARK_BEGIN(comp_d2h);
    hipMemcpy(nums, dev_nums, size, hipMemcpyDeviceToHost);
    CALI_MARK_END(comp_d2h);
    CALI_MARK_END(comm);

    hipFree(dev_nums);
}

bool confirm_sorted(float* nums) {
    float *dev_nums;
    bool *dev_sorted;
    bool sorted = true;
    size_t size = NUM_VALS * sizeof(float);

    hipMalloc((void**) &dev_nums, size);
    hipMalloc((void**) &dev_sorted, sizeof(bool));

    //MEM COPY FROM HOST TO DEVICE
    CALI_MARK_BEGIN(comm);
    CALI_MARK_BEGIN(correctness_h2d);
    hipMemcpy(dev_nums, nums, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_sorted, &sorted, sizeof(bool), hipMemcpyHostToDevice);
    CALI_MARK_END(correctness_h2d);
    CALI_MARK_END(comm);

    dim3 blocks(BLOCKS, 1);
    dim3 threads(THREADS, 1);

    // CHECKING CORRECTNESS
    CALI_MARK_BEGIN(correctness_check);
    confirm_sorted_step<<<blocks, threads>>>(dev_nums, NUM_VALS, dev_sorted);
    hipDeviceSynchronize();
    CALI_MARK_END(correctness_check);

    //MEM COPY FROM DEVICE TO HOST
    CALI_MARK_BEGIN(comm);
    CALI_MARK_BEGIN(correctness_d2h);
    hipMemcpy(&sorted, dev_sorted, sizeof(bool), hipMemcpyDeviceToHost);
    CALI_MARK_END(correctness_d2h);
    CALI_MARK_END(comm);

    hipFree(dev_nums);
    hipFree(dev_sorted);
    return sorted;
}

int main(int argc, char *argv[]) {
    CALI_CXX_MARK_FUNCTION;
    cali::ConfigManager mgr;
    mgr.start();

    // retrieve user input
    const char* input_type = argv[1];
    THREADS = atoi(argv[2]);
    NUM_VALS = atoi(argv[3]);
    BLOCKS = NUM_VALS / THREADS;

    printf("Number of threads: %d\n", THREADS);
    printf("Number of values: %d\n", NUM_VALS);
    printf("Number of blocks: %d\n\n", BLOCKS);

    // initialize array
    float *nums = (float*) malloc(NUM_VALS * sizeof(float));
    
    // fill array
    fill_array(nums, input_type);
    cout << "Data Initialized" << endl;

    // sort array
    bubble_sort(nums);
    cout << "Array Sorted" << endl;

    // check correctness
    if(confirm_sorted(nums)) {
        cout << "Correctness Check Passed!" << endl;
    }
    else {
        cout << "Correctness Check Failed..." << endl;
    }

    adiak::init(NULL);
    adiak::launchdate();    // launch date of the job
    adiak::libraries();     // Libraries used
    adiak::cmdline();       // Command line used to launch the job
    adiak::clustername();   // Name of the cluster
    adiak::value("Algorithm", "Odd Even Sort"); // The name of the algorithm you are using (e.g., "MergeSort", "BitonicSort")
    adiak::value("ProgrammingModel", "CUDA"); // e.g., "MPI", "CUDA", "MPIwithCUDA"
    adiak::value("Datatype", "float"); // The datatype of input elements (e.g., double, int, float)
    adiak::value("SizeOfDatatype", sizeof(float)); // sizeof(datatype) of input elements in bytes (e.g., 1, 2, 4)
    adiak::value("InputSize", NUM_VALS); // The number of elements in input dataset (1000)
    adiak::value("InputType", input_type); // For sorting, this would be "Sorted", "ReverseSorted", "Random", "1%perturbed"
    adiak::value("num_threads", NUM_VALS); // The number of CUDA or OpenMP threads
    adiak::value("num_blocks", BLOCKS); // The number of CUDA blocks 
    //adiak::value("group_num", group_number); // The number of your group (integer, e.g., 1, 10)
    adiak::value("implementation_source", "AI"); // Where you got the source code of your algorithm; choices: ("Online", "AI", "Handwritten").

    mgr.stop();
    mgr.flush();

    free(nums);
}
  