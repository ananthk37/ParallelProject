#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <cmath>
#include <stdio.h>
#include <stdlib.h>

#include <caliper/cali.h>
#include <caliper/cali-manager.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <adiak.hpp>

//number of bits to shift for each cut
#define SHIFT_NUMBER 4
//number of buckets created
#define NUM_BUCKETS 1 << SHIFT_NUMBER
//max distance to shift
#define MAX_SHIFT sizeof(unsigned int) * 8
//constant to & with to get right number of bits
#define LAST_DIGITS 0xF


using namespace std;


int THREADS;
int BLOCKS;
int NUM_VALS;

const char* data_init = "data_init";
const char* data_gen_h2d = "data_gen_h2d";
const char* data_gen_d2h = "data_gen_d2h";
const char* comp = "comp";
const char* comp_large = "comp_large";
const char* comm = "comm";
const char* comp_h2d = "comp_h2d";
const char* comp_d2h = "comp_d2h";
const char* correctness_check = "correctness_check";
const char* correctness_h2d = "correctness_h2d";
const char* correctness_d2h = "correctness_d2h";



__global__ void random_fill(unsigned int* nums, int size, const char* input_type) {
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    hiprandState state;
    hiprand_init(1, index, 0, &state);

    nums[index] = (unsigned int)hiprand(&state) * size;
}

__global__ void sorted_fill(unsigned int* nums, int size, const char* input_type) {
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    nums[index] = (unsigned int)index;
}

__global__ void reverse_fill(unsigned int* nums, int size, const char* input_type) {
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    nums[index] = (unsigned int)(size - index - 1);
}

__global__ void nearly_fill(unsigned int* nums, int size, const char* input_type) {
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    hiprandState state;
    hiprand_init(1, index, 0, &state);

    nums[index] = (unsigned int)(hiprand(&state) % blockDim.x) + blockIdx.x;
}


__global__ void confirm_sorted_step(unsigned int* nums, int size, bool* sorted) {
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    if(index < size - 1) {
        if(nums[index] > nums[index + 1]) {
            *sorted = false;
        }
    }
}

void fill_array(unsigned int* nums, const char* input_type) {
    unsigned int *dev_nums;
    size_t size = NUM_VALS * sizeof(unsigned int);
    
    hipMalloc((void**) &dev_nums, size);

    //MEM COPY FROM HOST TO DEVICE
    CALI_MARK_BEGIN(comm);
    CALI_MARK_BEGIN(data_gen_h2d);
    hipMemcpy(dev_nums, nums, size, hipMemcpyHostToDevice);
    CALI_MARK_END(data_gen_h2d);
    CALI_MARK_END(comm);

    dim3 blocks(BLOCKS, 1);
    dim3 threads(THREADS, 1);

    // FILLING ARRAY
    CALI_MARK_BEGIN(data_init);
    if(strcmp(input_type, "random") == 0) {
        random_fill<<<blocks, threads>>>(dev_nums, NUM_VALS, input_type);
    }
    if(strcmp(input_type, "sorted") == 0) {
        sorted_fill<<<blocks, threads>>>(dev_nums, NUM_VALS, input_type);
    }
    if(strcmp(input_type, "reverse") == 0) {
        reverse_fill<<<blocks, threads>>>(dev_nums, NUM_VALS, input_type);
    }
    if(strcmp(input_type, "nearly") == 0) {
        nearly_fill<<<blocks, threads>>>(dev_nums, NUM_VALS, input_type);
    }
    CALI_MARK_END(data_init);

    //MEM COPY FROM DEVICE TO HOST
    CALI_MARK_BEGIN(comm);
    CALI_MARK_BEGIN(data_gen_d2h);
    hipMemcpy(nums, dev_nums, size, hipMemcpyDeviceToHost);
    CALI_MARK_END(data_gen_d2h);
    CALI_MARK_END(comm);

    hipFree(dev_nums);
}

bool confirm_sorted(unsigned int* nums) {
    unsigned int *dev_nums;
    bool *dev_sorted;
    bool sorted = true;
    size_t size = NUM_VALS * sizeof(unsigned int);

    hipMalloc((void**) &dev_nums, size);
    hipMalloc((void**) &dev_sorted, sizeof(bool));

    //MEM COPY FROM HOST TO DEVICE
    CALI_MARK_BEGIN(comm);
    CALI_MARK_BEGIN(correctness_h2d);
    hipMemcpy(dev_nums, nums, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_sorted, &sorted, sizeof(bool), hipMemcpyHostToDevice);
    CALI_MARK_END(correctness_h2d);
    CALI_MARK_END(comm);

    dim3 blocks(BLOCKS, 1);
    dim3 threads(THREADS, 1);

    // CHECKING CORRECTNESS
    CALI_MARK_BEGIN(correctness_check);
    confirm_sorted_step<<<blocks, threads>>>(dev_nums, NUM_VALS, dev_sorted);
    hipDeviceSynchronize();
    CALI_MARK_END(correctness_check);

    //MEM COPY FROM DEVICE TO HOST
    CALI_MARK_BEGIN(comm);
    CALI_MARK_BEGIN(correctness_d2h);
    hipMemcpy(&sorted, dev_sorted, sizeof(bool), hipMemcpyDeviceToHost);
    CALI_MARK_END(correctness_d2h);
    CALI_MARK_END(comm);

    hipFree(dev_nums);
    hipFree(dev_sorted);
    return sorted;
}



__global__ void radix_sort_step(unsigned int* data, unsigned int* temp_data, unsigned int shift) {
    unsigned int id = blockId.x * blockDim.x + threadId.x;
    if(id < NUM_VALS) {
        unsigned int bucket = (data[id] >> shift) * LAST_DIGITS;
        unsigned int count = 0;


        __shared__ int histogram[NUM_BUCKETS];
        histogram[threadId.x] = 0;
        __synchthreads();

        for(int i = 0; i < NUM_VALS; i++) {
            histogram[bucket]++;
        }
        __synchthreads();

        for(int i = 0; i < bucket; i++) {
            count += histogram[i];
        }

        __synchthreads();

        temp_data[count + id] = data[id];

    }
}

void radix_sort(unsigned int* nums) {
    unsigned int* dev_nums, dev_temp;
    size_t size = NUM_VALS * sizeof(unsigned int);

    hipMalloc((void**)&dev_nums, size);
    hipMalloc((void**)&dev_temp, size);

    hipMemcpy(dev_nums, nums, size, hipMemcpyHostToDevice);

    for(unsigned int shift = 0; shift < MAX_SHIFT; shift += SHIFT_NUMBER) {
        radix_sort_step<<<(NUM_VALS + THREADS -)/THREADS, THREADS>>>(dev_nums, dev_temp, shift);
        hipMemcpy(nums, dev_temp, size, hipMemcpyDeviceToHost);

        int* temp = dev_nums;
        dev_nums = dev_temp;
        dev_temp = temp;
    }

    hipFree(dev_nums);
    hipFree(dev_temp);


}


int main(int argc, char *argv[]) {
    CALI_CXX_MARK_FUNCTION;
    cali::ConfigManager mgr;
    mgr.start();

    // retrieve user input
    const char* input_type = argv[1];
    THREADS = atoi(argv[2]);
    NUM_VALS = atoi(argv[3]);
    BLOCKS = NUM_VALS / THREADS;

    printf("Number of threads: %d\n", THREADS);
    printf("Number of values: %d\n", NUM_VALS);
    printf("Number of blocks: %d\n\n", BLOCKS);

    // initialize array
    unsigned int *nums = (unsigned int*) malloc(NUM_VALS * sizeof(unsigned int));
    
    // fill array
    fill_array(nums, input_type);
    cout << "Data Initialized" << endl;

    // sort array
    radix_sort(nums);
    cout << "Array Sorted" << endl;

    // check correctness
    if(confirm_sorted(nums)) {
        cout << "Correctness Check Passed!" << endl;
    }
    else {
        cout << "Correctness Check Failed..." << endl;
    }

    adiak::init(NULL);
    adiak::launchdate();    // launch date of the job
    adiak::libraries();     // Libraries used
    adiak::cmdline();       // Command line used to launch the job
    adiak::clustername();   // Name of the cluster
    adiak::value("Algorithm", "Radix Sort"); // The name of the algorithm you are using (e.g., "MergeSort", "BitonicSort")
    adiak::value("ProgrammingModel", "CUDA"); // e.g., "MPI", "CUDA", "MPIwithCUDA"
    adiak::value("Datatype", "unsigned int"); // The datatype of input elements (e.g., double, int, unsigned int)
    adiak::value("SizeOfDatatype", sizeof(unsigned int)); // sizeof(datatype) of input elements in bytes (e.g., 1, 2, 4)
    adiak::value("InputSize", NUM_VALS); // The number of elements in input dataset (1000)
    adiak::value("InputType", input_type); // For sorting, this would be "Sorted", "ReverseSorted", "Random", "1%perturbed"
    adiak::value("num_threads", NUM_VALS); // The number of CUDA or OpenMP threads
    adiak::value("num_blocks", BLOCKS); // The number of CUDA blocks 
    //adiak::value("group_num", group_number); // The number of your group (integer, e.g., 1, 10)
    adiak::value("implementation_source", "Online"); // Where you got the source code of your algorithm; choices: ("Online", "AI", "Handwritten").

    mgr.stop();
    mgr.flush();

    free(nums);
}
