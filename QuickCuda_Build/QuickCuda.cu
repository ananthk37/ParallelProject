#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <cmath>
#include <stdio.h>
#include <stdlib.h>

#include <caliper/cali.h>
#include <caliper/cali-manager.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <adiak.hpp>

using namespace std;

int THREADS;
int BLOCKS;
int NUM_VALS;

const char* data_init = "data_init";
const char* data_gen_h2d = "data_gen_h2d";
const char* data_gen_d2h = "data_gen_d2h";
const char* comp = "comp";
const char* comp_large = "comp_large";
const char* comm = "comm";
const char* comp_h2d = "comp_h2d";
const char* comp_d2h = "comp_d2h";
const char* correctness_check = "correctness_check";
const char* correctness_h2d = "correctness_h2d";
const char* correctness_d2h = "correctness_d2h";


__global__ void random_fill(int* nums, int size, const char* input_type) {
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    hiprandState state;
    hiprand_init(1, index, 0, &state);

    nums[index] = (int)hiprand_uniform(&state) * size;
}

__global__ void sorted_fill(int* nums, int size, const char* input_type) {
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    nums[index] = (int)index;
}

__global__ void reverse_fill(int* nums, int size, const char* input_type) {
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    nums[index] = (int)(size - index - 1);
}

__global__ void nearly_fill(int* nums, int size, const char* input_type) {
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    hiprandState state;
    hiprand_init(1, index, 0, &state);

    nums[index] = (int)hiprand_uniform(&state) * blockIdx.x;
}

void fill_array(int* nums, const char* input_type) {
    int *dev_nums;
    size_t size = NUM_VALS * sizeof(int);
    
    hipMalloc((void**) &dev_nums, size);

    //MEM COPY FROM HOST TO DEVICE
    CALI_MARK_BEGIN(comm);
    CALI_MARK_BEGIN(data_gen_h2d);
    hipMemcpy(dev_nums, nums, size, hipMemcpyHostToDevice);
    CALI_MARK_END(data_gen_h2d);
    CALI_MARK_END(comm);

    dim3 blocks(BLOCKS, 1);
    dim3 threads(THREADS, 1);

    // FILLING ARRAY
    CALI_MARK_BEGIN(data_init);
    if(strcmp(input_type, "random") == 0) {
        random_fill<<<blocks, threads>>>(dev_nums, NUM_VALS, input_type);
    }
    if(strcmp(input_type, "sorted") == 0) {
        sorted_fill<<<blocks, threads>>>(dev_nums, NUM_VALS, input_type);
    }
    if(strcmp(input_type, "reverse") == 0) {
        reverse_fill<<<blocks, threads>>>(dev_nums, NUM_VALS, input_type);
    }
    if(strcmp(input_type, "nearly") == 0) {
        nearly_fill<<<blocks, threads>>>(dev_nums, NUM_VALS, input_type);
    }
    CALI_MARK_END(data_init);

    //MEM COPY FROM DEVICE TO HOST
    CALI_MARK_BEGIN(comm);
    CALI_MARK_BEGIN(data_gen_d2h);
    hipMemcpy(nums, dev_nums, size, hipMemcpyDeviceToHost);
    CALI_MARK_END(data_gen_d2h);
    CALI_MARK_END(comm);

    hipFree(dev_nums);
}

__device__ int partition(int* dev_nums, int low, int high) {
    int pivot = dev_nums[high];
    int i = (low - 1);
  
    for (int j = low; j <= high - 1; j++) {
        if (dev_nums[j] < pivot) {
            i++;
            int temp = dev_nums[i];
            dev_nums[i] = dev_nums[j];
            dev_nums[j] = temp;
        }
    }
  
    int temp = dev_nums[i + 1];
    dev_nums[i + 1] = dev_nums[high];
    dev_nums[high] = temp;
  
    return (i + 1);
}

__global__ void quick_sort_step(int* dev_nums, int left, int right) {
    int stack[32];
    int top = -1;
  
    stack[++top] = left;
    stack[++top] = right;
  
    while (top >= 0) {
        right = stack[top--];
        left = stack[top--];
  
        int pivotIndex = partition(dev_nums, left, right);
  
        if (pivotIndex - 1 > left) {
            stack[++top] = left;
            stack[++top] = pivotIndex - 1;
        }
  
        if (pivotIndex + 1 < right) {
            stack[++top] = pivotIndex + 1;
            stack[++top] = right;
        }
    }
}

void quick_sort(int* nums) {

    int *dev_nums;
    size_t size = NUM_VALS * sizeof(int);
    
    hipMalloc((void**) &dev_nums, size);

    //MEM COPY FROM HOST TO DEVICE
    CALI_MARK_BEGIN(comm);
    CALI_MARK_BEGIN(comp_h2d);
    hipMemcpy(dev_nums, nums, size, hipMemcpyHostToDevice);
    CALI_MARK_END(comp_h2d);
    CALI_MARK_END(comm);

    dim3 blocks(BLOCKS, 1);
    dim3 threads(THREADS, 1);

    //QUICKSORT
    CALI_MARK_BEGIN(comp);
    CALI_MARK_BEGIN(comp_large);
    quick_sort_step<<<blocks, threads>>>(dev_nums, 0, NUM_VALS);
    CALI_MARK_END(comp_large);
    CALI_MARK_END(comp);

    //MEM COPY FROM DEVICE TO HOST
    CALI_MARK_BEGIN(comm);
    CALI_MARK_BEGIN(comp_d2h);
    hipMemcpy(nums, dev_nums, size, hipMemcpyDeviceToHost);
    CALI_MARK_END(comp_d2h);
    CALI_MARK_END(comm);

    hipFree(dev_nums);
}

__global__ void confirm_sorted_step(float* nums, int size, bool* sorted) {
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    if(index < size - 1) {
        if(nums[index] > nums[index + 1]) {
            *sorted = false;
        }
    }
}

bool confirm_sorted(int* nums) {
    float *dev_nums;
    bool *dev_sorted;
    bool sorted = true;
    size_t size = NUM_VALS * sizeof(int);

    hipMalloc((void**) &dev_nums, size);
    hipMalloc((void**) &dev_sorted, sizeof(bool));

    //MEM COPY FROM HOST TO DEVICE
    CALI_MARK_BEGIN(comm);
    CALI_MARK_BEGIN(correctness_h2d);
    hipMemcpy(dev_nums, nums, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_sorted, &sorted, sizeof(bool), hipMemcpyHostToDevice);
    CALI_MARK_END(correctness_h2d);
    CALI_MARK_END(comm);

    dim3 blocks(BLOCKS, 1);
    dim3 threads(THREADS, 1);

    // CHECKING CORRECTNESS
    CALI_MARK_BEGIN(correctness_check);
    confirm_sorted_step<<<blocks, threads>>>(dev_nums, NUM_VALS, dev_sorted);
    hipDeviceSynchronize();
    CALI_MARK_END(correctness_check);

    //MEM COPY FROM DEVICE TO HOST
    CALI_MARK_BEGIN(comm);
    CALI_MARK_BEGIN(correctness_d2h);
    hipMemcpy(&sorted, dev_sorted, sizeof(bool), hipMemcpyDeviceToHost);
    CALI_MARK_END(correctness_d2h);
    CALI_MARK_END(comm);

    hipFree(dev_nums);
    hipFree(dev_sorted);
    return sorted;
}

int main(int argc, char *argv[]) {
    CALI_CXX_MARK_FUNCTION;
    cali::ConfigManager mgr;
    mgr.start();

    // retrieve user input
    const char* input_type = argv[1];
    THREADS = atoi(argv[2]);
    NUM_VALS = atoi(argv[3]);
    BLOCKS = NUM_VALS / THREADS;

    printf("Number of threads: %d\n", THREADS);
    printf("Number of values: %d\n", NUM_VALS);
    printf("Number of blocks: %d\n\n", BLOCKS);

    // initialize array
    int *nums = (int*) malloc(NUM_VALS * sizeof(int));
    
    // fill array
    fill_array(nums, input_type);
    cout << "Data Initialized" << endl;

    // sort array
    quick_sort(nums);
    cout << "Array Sorted" << endl;

    // check correctness
    if(confirm_sorted(nums)) {
        cout << "Correctness Check Passed!" << endl;
    }
    else {
        cout << "Correctness Check Failed..." << endl;
    }

    adiak::init(NULL);
    adiak::launchdate();    // launch date of the job
    adiak::libraries();     // Libraries used
    adiak::cmdline();       // Command line used to launch the job
    adiak::clustername();   // Name of the cluster
    adiak::value("Algorithm", "QuickSort"); // The name of the algorithm you are using (e.g., "MergeSort", "BitonicSort")
    adiak::value("ProgrammingModel", "CUDA"); // e.g., "MPI", "CUDA", "MPIwithCUDA"
    adiak::value("Datatype", "int"); // The datatype of input elements (e.g., double, int, float)
    adiak::value("SizeOfDatatype", sizeof(int)); // sizeof(datatype) of input elements in bytes (e.g., 1, 2, 4)
    adiak::value("InputSize", NUM_VALS); // The number of elements in input dataset (1000)
    adiak::value("InputType", input_type); // For sorting, this would be "Sorted", "ReverseSorted", "Random", "1%perturbed"
    adiak::value("num_threads", NUM_VALS); // The number of CUDA or OpenMP threads
    adiak::value("num_blocks", BLOCKS); // The number of CUDA blocks 
    adiak::value("group_num", 3); // The number of your group (integer, e.g., 1, 10)
    adiak::value("implementation_source", "AI"); // Where you got the source code of your algorithm; choices: ("Online", "AI", "Handwritten").

    mgr.stop();
    mgr.flush();

    free(nums);
}
  